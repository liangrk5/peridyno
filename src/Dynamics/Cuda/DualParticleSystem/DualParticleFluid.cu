#include "hip/hip_runtime.h"
#include "DualParticleFluid.h"
//DataType
#include "Auxiliary/DataSource.h"

//Collision
#include "Collision/NeighborPointQuery.h"

//ParticleSystem
#include "ParticleSystem/Module/ImplicitViscosity.h"
#include "ParticleSystem/Module/ParticleIntegrator.h"
#include "ParticleSystem/Module/SemiImplicitDensitySolver.h"
#include "ParticleSystem/Module/VariationalApproximateProjection.h"

//DualParticleSystem
#include "Module/DualParticleIsphModule.h"
#include <DualParticleSystem/Module/ThinFeature.h>


namespace dyno
{
	__global__ void  DPS_AttributeReset(
		DArray<Attribute> att
	)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= att.size()) return;

		att[pId].setFluid();
		att[pId].setDynamic();
	}

	template<typename TDataType>
	DualParticleFluid<TDataType>::DualParticleFluid()
		: DualParticleFluid<TDataType>::DualParticleFluid(3)
	{

	}


	template<typename TDataType>
	DualParticleFluid<TDataType>::DualParticleFluid(int key)
		: ParticleFluid<TDataType>()
	{
		this->varVirtualParticleSamplingStrategy()->getDataPtr()->setCurrentKey(key);
		this->varReshuffleParticles()->setValue(false);
		this->varSmoothingLength()->setValue(2.4);

		this->animationPipeline()->clear();

		auto m_nbrQuery = std::make_shared<NeighborPointQuery<TDataType>>();
		this->stateSmoothingLength()->connect(m_nbrQuery->inRadius());
		this->statePosition()->connect(m_nbrQuery->inPosition());
		this->animationPipeline()->pushModule(m_nbrQuery);

		if (key == EVirtualParticleSamplingStrategy::SpatiallyAdaptiveStrategy)
		{
			auto m_adaptive_virtual_position = std::make_shared<VirtualSpatiallyAdaptiveStrategy<TDataType>>();
			this->statePosition()->connect(m_adaptive_virtual_position->inRPosition());
			m_adaptive_virtual_position->varSamplingDistance()->setValue(Real(0.005));		/**Virtual particle radius*/
			m_adaptive_virtual_position->varCandidatePointCount()->getDataPtr()->setCurrentKey(VirtualSpatiallyAdaptiveStrategy<TDataType>::neighbors_33);
			vpGen = m_adaptive_virtual_position;
		}
		else if (key == EVirtualParticleSamplingStrategy::ParticleShiftingStrategy)
		{
			auto m_virtual_particle_shifting = std::make_shared<VirtualParticleShiftingStrategy<TDataType >>();
			this->stateFrameNumber()->connect(m_virtual_particle_shifting->inFrameNumber());
			this->stateFrameNumber()->connect(m_virtual_particle_shifting->inFrameNumber());
			this->statePosition()->connect(m_virtual_particle_shifting->inRPosition());
			this->stateTimeStep()->connect(m_virtual_particle_shifting->inTimeStep());
			this->animationPipeline()->pushModule(m_virtual_particle_shifting);
			vpGen = m_virtual_particle_shifting;
		}
		else if (key == EVirtualParticleSamplingStrategy::ColocationStrategy)
		{
			auto m_virtual_equal_to_Real = std::make_shared<VirtualColocationStrategy<TDataType>>();
			this->statePosition()->connect(m_virtual_equal_to_Real->inRPosition());
			this->animationPipeline()->pushModule(m_virtual_equal_to_Real);
			vpGen = m_virtual_equal_to_Real;
		}
		else if (key == EVirtualParticleSamplingStrategy::FissionFusionStrategy)
		{
			auto feature = std::make_shared<ThinFeature<TDataType>>();
			this->statePosition()->connect(feature->inPosition());
			m_nbrQuery->outNeighborIds()->connect(feature->inNeighborIds());
			this->stateSmoothingLength()->connect(feature->inSmoothingLength());
			this->stateSamplingDistance()->connect(feature->inSamplingDistance());
			feature->varThreshold()->setValue(0.1f);
			this->animationPipeline()->pushModule(feature);

			auto gridFission = std::make_shared<VirtualFissionFusionStrategy<TDataType>>();
			gridFission->varTransitionRegionThreshold()->setValue(0.01);
			feature->outThinSheet()->connect(gridFission->inThinSheet());
			feature->outThinFeature()->connect(gridFission->inThinFeature());
			this->statePosition()->connect(gridFission->inRPosition());
			this->stateVelocity()->connect(gridFission->inRVelocity());
			m_nbrQuery->outNeighborIds()->connect(gridFission->inNeighborIds());
			this->stateSmoothingLength()->connect(gridFission->inSmoothingLength());
			this->stateSamplingDistance()->connect(gridFission->inSamplingDistance());
			this->stateFrameNumber()->connect(gridFission->inFrameNumber());
			this->stateTimeStep()->connect(gridFission->inTimeStep());
			this->animationPipeline()->pushModule(gridFission);
			gridFission->varMinDist()->setValue(0.002);
			vpGen = gridFission;
		}

		this->animationPipeline()->pushModule(vpGen);
		vpGen->outVirtualParticles()->connect(this->stateVirtualPosition());

		auto m_rv_nbrQuery = std::make_shared<NeighborPointQuery<TDataType>>();
		this->stateSmoothingLength()->connect(m_rv_nbrQuery->inRadius());
		this->statePosition()->connect(m_rv_nbrQuery->inOther());
		vpGen->outVirtualParticles()->connect(m_rv_nbrQuery->inPosition());
		this->animationPipeline()->pushModule(m_rv_nbrQuery);

		auto m_vr_nbrQuery = std::make_shared<NeighborPointQuery<TDataType>>();
		this->stateSmoothingLength()->connect(m_vr_nbrQuery->inRadius());
		this->statePosition()->connect(m_vr_nbrQuery->inPosition());
		vpGen->outVirtualParticles()->connect(m_vr_nbrQuery->inOther());
		this->animationPipeline()->pushModule(m_vr_nbrQuery);

		auto m_vv_nbrQuery = std::make_shared<NeighborPointQuery<TDataType>>();
		this->stateSmoothingLength()->connect(m_vv_nbrQuery->inRadius());
		vpGen->outVirtualParticles()->connect(m_vv_nbrQuery->inPosition());
		this->animationPipeline()->pushModule(m_vv_nbrQuery);

		auto m_dualIsph = std::make_shared<DualParticleIsphModule<TDataType>>();
		this->stateSmoothingLength()->connect(m_dualIsph->varSmoothingLength());
		this->stateSamplingDistance()->connect(m_dualIsph->varSamplingDistance());
		this->stateTimeStep()->connect(m_dualIsph->inTimeStep());
		this->statePosition()->connect(m_dualIsph->inRPosition());
		vpGen->outVirtualParticles()->connect(m_dualIsph->inVPosition());
		this->stateVelocity()->connect(m_dualIsph->inVelocity());
		m_dualIsph->varResidualThreshold()->setValue(0.001f);
		//this->stateParticleAttribute()->connect(m_dualIsph->inParticleAttribute());
		//this->stateBoundaryNorm()->connect(m_dualIsph->inBoundaryNorm());
		m_nbrQuery->outNeighborIds()->connect(m_dualIsph->inNeighborIds());
		m_rv_nbrQuery->outNeighborIds()->connect(m_dualIsph->inRVNeighborIds());
		m_vr_nbrQuery->outNeighborIds()->connect(m_dualIsph->inVRNeighborIds());
		m_vv_nbrQuery->outNeighborIds()->connect(m_dualIsph->inVVNeighborIds());
		m_dualIsph->varWarmStart()->setValue(true);
		this->animationPipeline()->pushModule(m_dualIsph);

		auto m_integrator = std::make_shared<ParticleIntegrator<TDataType>>();
		this->stateTimeStep()->connect(m_integrator->inTimeStep());
		this->statePosition()->connect(m_integrator->inPosition());
		this->stateVelocity()->connect(m_integrator->inVelocity());
		this->animationPipeline()->pushModule(m_integrator);

		auto m_visModule = std::make_shared<ImplicitViscosity<TDataType>>();
		m_visModule->varViscosity()->setValue(Real(0.5));
		this->stateTimeStep()->connect(m_visModule->inTimeStep());
		this->stateSamplingDistance()->connect(m_visModule->inSamplingDistance());
		this->stateSmoothingLength()->connect(m_visModule->inSmoothingLength());
		this->stateTimeStep()->connect(m_visModule->inTimeStep());
		this->statePosition()->connect(m_visModule->inPosition());
		this->stateVelocity()->connect(m_visModule->inVelocity());
		m_nbrQuery->outNeighborIds()->connect(m_visModule->inNeighborIds());
		this->animationPipeline()->pushModule(m_visModule);
	}


	template<typename TDataType>
	DualParticleFluid<TDataType>::~DualParticleFluid()
	{
	
	}

	template<typename TDataType>
	void DualParticleFluid<TDataType>::resetStates()
	{
		this->ParticleFluid<TDataType>::resetStates();

		auto ptSet = this->statePointSet()->getDataPtr();
		if(ptSet != nullptr)
		{
			auto pts = ptSet->getPoints();
		}

		std::cout << "Real particle number " << this->statePosition()->size() << std::endl;

		if (this->stateVirtualPointSet()->isEmpty())
		{
			this->stateVirtualPointSet()->allocate();
		}

		if (!this->stateVirtualPosition()->isEmpty())
		{
			auto virtualPoints = this->stateVirtualPointSet()->getDataPtr();
			virtualPoints->setPoints(this->stateVirtualPosition()->getData());
		}
		else
		{
			auto virtualPoints = this->stateVirtualPointSet()->getDataPtr();
			virtualPoints->clear();
		}
	}

	template<typename TDataType>
	void DualParticleFluid<TDataType>::preUpdateStates()
	{
		this->varReshuffleParticles()->setValue(false);
		this->ParticleFluid<TDataType>::preUpdateStates();

	}


	template<typename TDataType>
	void DualParticleFluid<TDataType>::postUpdateStates()
	{
		this->ParticleSystem<TDataType>::postUpdateStates();

		if (!this->stateVirtualPosition()->isEmpty())
		{
			auto virtualPoints = this->stateVirtualPointSet()->getDataPtr();
			virtualPoints->setPoints(this->stateVirtualPosition()->getData());
		}
		else
		{
			auto virtualPoints = this->stateVirtualPointSet()->getDataPtr();
			virtualPoints->clear();
		}
	}
	
	DEFINE_CLASS(DualParticleFluid);
}


