#include "hip/hip_runtime.h"
#include "RigidWaterCoupling.h"

#include "Math/Lerp.h"

#include "Primitive/Primitive3D.h"

namespace dyno
{
	template<typename TDataType>
	RigidWaterCoupling<TDataType>::RigidWaterCoupling()
		: Node()
	{
	}

	template<typename TDataType>
	RigidWaterCoupling<TDataType>::~RigidWaterCoupling()
	{
	}

	template<typename TDataType>
	void RigidWaterCoupling<TDataType>::resetStates()
	{
	}

	template<typename Coord, typename Triangle>
	__global__ void C_ComputeForceAndTorque(
		DArray<Coord> force,
		DArray<Coord> torque,
		DArray<Coord> vertices,
		DArray<Triangle> indices,
		DArray2D<Real> heights,
		Coord barycenter,
		Coord gravity,
		Coord origin,
		Real waterLevel,
		Real spacing,
		Real rho)
	{
		int tId = threadIdx.x + blockIdx.x * blockDim.x;
		if (tId >= indices.size()) return;

		Triangle index_i = indices[tId];

		Coord v0 = vertices[index_i[0]];
		Coord v1 = vertices[index_i[1]];
		Coord v2 = vertices[index_i[2]];

		Triangle3D triangle(v0, v1, v2);

		//Triangle normal
		Coord normal_i = (v2 - v0).cross(v1 - v0);
		normal_i.normalize();

		Coord triangle_center = (v0 + v1 + v2) / Real(3);

		//Calculate buoyancy
		Real sea_level = bilinear(heights, (triangle_center.x - origin.x) / spacing, (triangle_center.z - origin.z) / spacing) + waterLevel;
		Real h = triangle_center.y < sea_level ? (sea_level - triangle_center.y) : Real(0);

		Real pressure = rho * gravity.norm() * h;

		Coord force_i = pressure * triangle.area() * normal_i;
		Coord torque_i = -force_i.cross(triangle_center - barycenter);

		force[tId] = force_i;
		torque[tId] = torque_i;
	}

	template<typename TDataType>
	void RigidWaterCoupling<TDataType>::updateStates()
	{
		Real dt = this->stateTimeStep()->getData();

		auto vessels = this->getVessels();
		auto ocean = this->getOcean();

		auto patch = ocean->getOceanPatch();

		Real waterLevel = ocean->varWaterLevel()->getValue();

		for (auto mesh : vessels)
		{
			auto& triangles = mesh->stateEnvelope()->getData();

			Real mass = mesh->stateMass()->getData();
			Coord barycenter = mesh->stateBarycenter()->getData();
			Coord velocity = mesh->stateVelocity()->getData();
			Coord angular_velocity = mesh->stateAngularVelocity()->getData();
			Matrix inertia = mesh->stateInertia()->getData();

			Coord gravity = mesh->varGravity()->getData();

			auto& vertices = triangles.getPoints();
			auto& indices = triangles.triangleIndices();

			uint num = indices.size();

			if (mForce.size() != num) {
				mForce.resize(num);
				mTorque.resize(num);
			}

			auto heights = patch->stateHeightField()->getDataPtr();
			auto& displacements = heights->calculateHeightField();
			Coord origin = heights->getOrigin();
			Real h = heights->getGridSpacing();

			cuExecute(num,
				C_ComputeForceAndTorque,
				mForce,
				mTorque,
				vertices,
				indices,
				displacements,
				barycenter,
				gravity,
				origin,
				waterLevel,
				h,
				Real(1000));

			Coord F_total = mReduce.accumulate(mForce.begin(), mForce.size());
			Coord T_total = mReduce.accumulate(mTorque.begin(), mTorque.size());

			velocity += dt * F_total / mass;
			angular_velocity += dt * inertia.inverse() * T_total;

			velocity *= this->varDamping()->getValue();
			angular_velocity *= this->varRotationalDamping()->getValue();

			mesh->stateVelocity()->setValue(velocity);
			mesh->stateAngularVelocity()->setValue(angular_velocity);
		}
	}

	DEFINE_CLASS(RigidWaterCoupling);
}
